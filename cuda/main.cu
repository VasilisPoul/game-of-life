#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <assert.h>
#include <time.h>

/*COLOR*/
#define RED "\x1B[31m"
#define BLUE "\x1B[34m"

/*BOLD-COLOR*/
#define B_RED "\x1B[1m\x1B[31m"
#define B_GREEN "\x1B[1m\x1B[32m"

/*RESET COLOR*/
#define RESET  "\x1B[0m"

#define N 16
#define M 4
#define FILE_NAME "/home/msi/projects/CLionProjects/game-of-life/cuda/test-files/64x64.txt"
#define STEPS 1

int **allocate2DArray(int rows, int columns) {
    int **block;
    int i;
    block = (int **) malloc(rows * sizeof(int *));
    block[0] = (int *) malloc(rows * columns * sizeof(int));
    for (i = 1; i < rows; i++) {
        block[i] = &(block[0][i * rows]);
    }
    memset(block[0], 0, rows * columns * sizeof(int));
    return block;
}

void free2DArray(int **block) {
    free(block[0]);
    free(block);
}

void print_array(int **array, bool split, bool internals, int rowDim, int colDim, int localRowDim, int localColDim) {
    printf("\n");
    for (int i = 0; i < rowDim; i++) {
        for (int j = 0; j < colDim; j++) {
            if ((rowDim != localRowDim && colDim != localColDim)) {
//                printf("%s %c ", array[i][j] == '1' ? RED"\u2B1B" RESET : "\u2B1C",
//                       (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                if (array[i][j]) {
                    printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                } else {
                    printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                }
            } else {
                if ((i == 0 || i == rowDim - 1) || (j == 0 || j == colDim - 1)) {
//                    printf("%s %c ", array[i][j] == '1' ? B_GREEN"\u2B1B" RESET : "\u2B1C",
//                           (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    if (array[i][j]) {
                        printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    } else {
                        printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    }
                } else if (internals && ((i == 1 || i == rowDim - 2) || (j == 1 || j == colDim - 2))) {
//                    printf("%s %c ", array[i][j] == '1' ? BLUE"\u2B1B" RESET : "\u2B1C",
//                           (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    if (array[i][j]) {
                        printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    } else {
                        printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    }
                } else {
//                    printf("%s %c ", array[i][j] == '1' ? RED"\u2B1B" RESET : "\u2B1C",
//                           (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    if (array[i][j]) {
                        printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    } else {
                        printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    }
                }
            }
        }
        printf("\n%c", (split && (i + 1) % localRowDim == 0) ? '\n' : '\0');
    }
    printf("\n");
}

// Device code
__global__ void kernel(int *old, int *current) {
    __shared__ int local[M + 2][M + 2];
    unsigned int local_row = threadIdx.x;
    unsigned int local_col = threadIdx.y;
    unsigned int local_thread_id = local_col + local_row * M;

    unsigned int ix = blockIdx.x * (blockDim.x) + threadIdx.x;
    unsigned int iy = blockIdx.y * (blockDim.y) + threadIdx.y;
    unsigned int idx = ix * N + iy;

    // // split internals
    // if (blockIdx.x > 0 && blockIdx.x < gridDim.x - 1 && blockIdx.y > 0 && blockIdx.y < gridDim.y - 1) {
    //     local[local_row + 1][local_col + 1] = old[idx];

    //     //up
    //     if (local_row == 0) {
    //         local[local_row][local_col + 1] = old[idx - N];
    //     }

    //     //down
    //     if (local_row == blockDim.x - 2) {
    //         local[local_row + 3][local_col + 1] = old[idx + 2 * N];
    //     }

    //     //left
    //     if (local_col == 0) {
    //         local[local_row + 1][local_col] = old[idx - 1];
    //     }

    //     //right
    //     if (local_col == blockDim.y - 2) {
    //         local[local_row + 1][local_col + 3] = old[idx + 2];
    //     }

    //     //up left
    //     if (local_col == 0 && local_row == 0) {
    //         local[local_row][local_col] = old[idx - N - 1];
    //     }

    //     //up right
    //     if (local_col == blockDim.y - 2 && local_row == 0) {
    //         local[local_row][local_col + 3] = old[idx - N + 2];
    //     }

    //     //down left
    //     if (local_col == 0 && local_row == blockDim.y - 2) {
    //         local[local_row + 3][local_col] = old[idx + 2 * N - 1];
    //     }

    //     //down right
    //     if (local_col == blockDim.y - 2 && local_row == blockDim.x - 2) {
    //         local[local_row + 3][local_col + 3] = old[idx + 2 * N + 2];
    //     }
    // } else {
        // // Todo: calculate external blocks


        if(blockIdx.x == 0){
            // //internals
            // local[local_row + 1][local_col + 1] = old[idx];

            // //up
            // if (local_row == 0){
            //     local[local_row][local_col + 1] = old[idx + (N-1) * N];
            // }
            // //down idio me ta apo panw 
            // if (local_row == blockDim.x - 2){
            //     local[local_row + 3][local_col + 1] = old[idx + 2 * N];
            // }
            // //left
            // if (local_col == 0){
            //     //full aristera
            //     if (blockIdx.y == 0){
            //         local[local_row + 1][local_col] = old[idx + N - 1];
            //     }
            //     else{
            //         local[local_row + 1][local_col] = old[idx - 1];
            //     }
            // }
            // //right
            // if(local_col == blockDim.y-1){
                
            //     if (blockIdx.y != gridDim.y - 1){
                    
            //         local[local_row + 1][local_col+2] = old[idx + 1];  
            //     }
            //     //full deksia
            //     else{
            //         printf("Hey!!!\n");
            //         local[local_row + 1][local_col + 2] = old[idx - N+1];
                    
            //         printf("local_row+1: %d, local_col+2: %d\n", local_row+1, local_col+2);
            //         printf("old[idx + 1]: %5.4d\n", old[idx + 1]); 
            //     }
            // }
            // //up left
            // if(local_col == 0 && local_row == 0){
            //     //terma aristera block
            //     if (blockIdx.y == 0){
            //         local[local_row][local_col] = old[idx + N * N - 1];
            //     }
            //     else{
                    
            //         local[local_row][local_col] = old[idx + (N-1)*N - 1];
            //     }
            // }
            // //up right
            // if (local_row == 0 && local_col == blockDim.y - 2){
            //     if(blockIdx.y != gridDim.y - 1){
            //         local[local_row][local_col + 3] = old[idx + (N-1)*N + 2]; 
            //     }
            //     //terma deksia
            //     else{
            //         local[local_row][local_col + 3] = old[idx + (N-1)*N- N+2 ];
            //     }
            // }
            // //down left
            // if(local_row == blockDim.x - 2 && local_col == 0){
            //     //terma aristera block
            //     if (blockIdx.y == 0){
            //         local[local_row+3][local_col] = old[idx + 3 * N - 1];
            //     }
            //     else {
            //         local[local_row+3][local_col] = old[idx + 3 * N - 1 - N];
            //     }
            // }
            // //down right
            // if (local_row == blockDim.x-2 && local_col == blockDim.y - 2){
            //     if(blockIdx.y != gridDim.y - 1){
            //         local[local_row + 3][local_col + 3] = old[idx + 2*N +2];   
            //     }
            //     //terma deksia
            //     else {
            //         //TODO
            //         local[local_row + 3][local_col + 3] = old[idx + N +2];
            //         // printf("local_row+3: %d, local_col+3: %d\n", local_row+3, local_col+3);
            //         // printf("old[idx + N +2]: %5.4d\n", old[idx + N +2]); 
            //     }
            // }            
        }


        if (blockIdx.x == gridDim.x - 1 ) {
            // local[local_row + 1][local_col + 1] = old[idx];

            // //up
            // if (local_row == 0){
            //     //idio me apo panw
            //     local[local_row][local_col + 1] = old[idx - N];
            // }
            // //down idio me ta apo panw 
            // if (local_row == blockDim.x - 1){
            //     local[local_row + 2][local_col + 1] = old[idx - N*(N-1)]; 
            // }
            // //left
            // if (local_col == 0){
            //     //full aristera
            //     if (blockIdx.y == 0){
            //         local[local_row + 1][local_col] = old[idx + N - 1];
            //         // printf("local_row+1: %d, local_col: %d\n", local_row+1, local_col);
            //         // printf("old[idx + N - 1]: %5.4d\n", old[idx + N - 1]); 
            //     }
            //     else{
            //         local[local_row + 1][local_col] = old[idx - 1];
            //     }
            // }
            // //right
            // if(local_col == blockDim.y-1){
                
            //     if (blockIdx.y != gridDim.y - 1){
                    
            //         local[local_row + 1][local_col+2] = old[idx + 1];  
            //     }
            //     //full deksia
            //     else{
            //         local[local_row + 1][local_col + 2] = old[idx - N+1];
                     
            //     }
            // }
            // //up left
            // if(local_col == 0 && local_row == 0){
            //     //terma aristera block
            //     if (blockIdx.y == 0){
            //         local[local_row][local_col] = old[idx -1];
            //     }
            //     else{
            //         local[local_row][local_col] = old[idx - N -1];
            //     }
            // }
            // //up right
            // if (local_row == 0 && local_col == blockDim.y - 1){
            //     if(blockIdx.y != gridDim.y - 1){
            //         local[local_row][local_col + 2] = old[idx - N + 1]; 

            //     }
            //     //terma deksia
            //     else{
            //         //TODO
            //         local[local_row][local_col + 2] = old[idx- 2*N +1];
            //     }
            // }
            // //down left
            // if(local_row == blockDim.x - 1 && local_col == 0){
            //     //terma aristera block
            //     if (blockIdx.y == 0){
            //         local[local_row+2][local_col] = old[idx - (N-1) * (N-1) ];  
            //     }
                
            //     else {
            //         local[local_row+2][local_col] = old[idx - N * (N-1)-1];

            //         printf("local_row+2: %d, local_col: %d\n", local_row+2, local_col);
            //         printf("old[idx-N-1]: %5.4d\n", old[idx - N * (N-1)-1]);
            //     }
            // }
            // //down right
            // if (local_row == blockDim.x-1 && local_col == blockDim.y - 1){
            //     if(blockIdx.y != gridDim.y - 1){
                    
            //         local[local_row+2][local_col+2] = old[idx - (N-1) * N + 1];
            //     }
            //     else if (blockIdx.y == gridDim.y - 1){
            //         local[local_row+2][local_col+2] = old[idx -(N-1)*N + 1 - N];

            //         printf("local_row+2: %d, local_col: %d\n", local_row+2, local_col);
            //         printf("old[idx-N-1]: %5.4d\n", old[idx -(N-1)*N + 1 - N]);
            //     }
               
            // }            


        }

        old[idx] = idx;

        if (blockIdx.x == 1 && blockIdx.y == 0){
            local[local_row + 1][local_col + 1] = old[idx];

                //up
                if (local_row == 0) {
                    local[local_row][local_col + 1] = old[idx - N];
                }

                //down
                if (local_row == blockDim.x - 2) {
                    local[local_row + 3][local_col + 1] = old[idx + 2 * N];
                }

                //right
                if (local_col == blockDim.y - 1) {
                    local[local_row + 1][local_col + 2] = old[idx + 1];
                }
                //left
                if (local_col == 0) {
                    local[local_row + 1][local_col] = old[idx + N - 1];
                }
                //up right
                if (local_col == blockDim.y - 2 && local_row == 0) {
                    local[local_row][local_col + 3] = old[idx - N + 2];
                }
                //down right
                if (local_col == blockDim.y - 2 && local_row == blockDim.x - 2) {
                    local[local_row + 3][local_col + 3] = old[idx + 2 * N + 2];
                }
                //up left
                if(local_col == 0 && local_row == 0){
                    //terma aristera block
                    if (blockIdx.y == 0){
                        local[local_row][local_col] = old[idx -1];
                    }
                
                }

                //down left
                if(local_row == blockDim.x - 2 && local_col == 0){
                    //terma aristera block
                    if (blockIdx.y == 0){
                        local[local_row+3][local_col] = old[idx + 3 * N - 1];
                    }
                }
                


        }
        

        __syncthreads();
       //print block
       if (ix == 4 && iy == 0) {
           for (int i = 0; i < M + 2; i++) {
               for (int j = 0; j < M + 2; j++) {
                    if (i > 0 && i < M+1 && j > 0 && j < M+1){
                        printf(RED"%5.4d " RESET, local[i][j]);
                    }else{
                        printf("%5.4d ", local[i][j]);
                    }
                 
               }
               printf("\n");
           }
           printf("\n");
       }


//    }


    __syncthreads();

    //Todo: Calculate cells

}

// Host code
int main() {
    int **host_array = nullptr, *device_old = nullptr, *device_current = nullptr, *temp = nullptr;;
    int i = 0, fd = 0;
    double time_spent = 0.0;
    clock_t begin, end;

    // Threads (2D) per block
    dim3 m(M, M);

    // Blocks (2D grid)
    dim3 n((unsigned int) ((N + (float) M - 1) / (float) M), (unsigned int) ((N + (float) M - 1) / (float) M));

    //assert(N == M * M * 2);

    // Array allocations
    host_array = allocate2DArray(N, N);

//    // Read file
//    if ((fd = open(FILE_NAME, O_RDONLY)) < 0) {
//        fprintf(stderr, "Could not open file \"%s\"\n", FILE_NAME);
//        return -1;
//    }
//    i = 1;
//    while (read(fd, &host_array[i++][1], N));
//    close(fd);

    printf("host_array before:\n");
    print_array(host_array, true, true, N, N, N, N);

    // Initialize 2D 'old' array on device
    hipMalloc((void **) &device_old, N * N * sizeof(int));

    // Copy 2D 'old' array on device
    hipMemcpy(device_old, host_array[0], N * N * sizeof(int), hipMemcpyHostToDevice);

    // Initialize 2D 'current' array on device
    hipMalloc((void **) &device_current, N * N * sizeof(int));

    // Copy 2D 'current' array on device
    hipMemset(device_current, '0', N * N * sizeof(int));

    begin = clock();

    // Computations
    for (i = 0; i < STEPS; i++) {
        // Call device function
        kernel<<<n, m>>>(device_old, device_current);

        // Copy 2D 'device_current' array on host
        hipMemcpy(host_array[0], device_old, sizeof(int) * N * N, hipMemcpyDeviceToHost);

        printf("host_array on step %d:\n", i);
        print_array(host_array, true, true, N, N, N, N);

        // Swap 'device_old' and 'device_current' arrays
        temp = device_old;
        device_old = device_current;
        device_current = temp;
    }

    end = clock();

    time_spent = (double) (end - begin) / CLOCKS_PER_SEC;

    printf("time_spent=%f\n", time_spent);

    // Free memory
    hipFree(device_old);
    hipFree(device_current);
    free2DArray(host_array);
    return 0;
}
